/*
* (c) 2015 Virginia Polytechnic Institute & State University (Virginia Tech)
*
*   This program is free software: you can redistribute it and/or modify
*   it under the terms of the GNU General Public License as published by
*   the Free Software Foundation, version 2.1
*
*   This program is distributed in the hope that it will be useful,
*   but WITHOUT ANY WARRANTY; without even the implied warranty of
*   MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
*   GNU General Public License, version 2.1, for more details.
*
*   You should have received a copy of the GNU General Public License
*
*/

#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include <vector>
#include <algorithm>

#include "bb_segsort.cuh"

using std::vector;
using std::pair;

using index_t = int;
using seg_t = int;

#define CUDA_CHECK(_e, _s) if(_e != hipSuccess) { \
        std::cout << "CUDA error (" << _s << "): " << hipGetErrorString(_e) << std::endl; \
        return 0; }

template<class K, class T>
void gold_segsort(vector<K> &key, vector<T> &val, index_t n, const vector<seg_t> &seg, index_t m);

int show_mem_usage();

int main(int argc, char **argv)
{
    using key_t = int;
    using val_t = double;

    hipError_t err;
    // index_t n = 400 000 000;
    index_t n = 1UL << 24;
    vector<key_t> key(n, 0);
    vector<val_t> val(n, 0.0);
    for(auto &k: key)
        k = rand()%(n-1-0+1)+0;
    for(auto &v: val)
        v = (val_t)(rand()%(n-1-0+1)+0);
    seg_t max_seg_sz = 10000;
    seg_t min_seg_sz = 0;
    vector<seg_t> seg;
    seg_t off = 0;
    seg.push_back(off); // must have a zero
    while(off < n)
    {
        seg.push_back(off);
        seg_t sz = rand()%(max_seg_sz-min_seg_sz+1)+min_seg_sz;
        off = seg.back()+sz;
    }
    index_t m = seg.size();
    printf("synthesized segments # %d (max_size: %d, min_size: %d)\n", m, max_seg_sz, min_seg_sz);

    // cout << "key:\n"; for(auto k: key) cout << k << ", "; cout << endl;
    // cout << "val:\n"; for(auto v: val) cout << v << ", "; cout << endl;
    // cout << "seg:\n"; for(auto s: seg) cout << s << ", "; cout << endl;

    key_t *key_d;
    val_t *val_d;
    key_t *seg_d;
    err = hipMalloc((void**)&key_d, sizeof(key_t)*n);
    CUDA_CHECK(err, "alloc key_d");
    err = hipMalloc((void**)&val_d, sizeof(val_t)*n);
    CUDA_CHECK(err, "alloc val_d");
    err = hipMalloc((void**)&seg_d, sizeof(key_t)*n);
    CUDA_CHECK(err, "alloc seg_d");

    err = hipMemcpy(key_d, &key[0], sizeof(key_t)*n, hipMemcpyHostToDevice);
    CUDA_CHECK(err, "copy to key_d");
    err = hipMemcpy(val_d, &val[0], sizeof(val_t)*n, hipMemcpyHostToDevice);
    CUDA_CHECK(err, "copy to val_d");
    err = hipMemcpy(seg_d, &seg[0], sizeof(key_t)*m, hipMemcpyHostToDevice);
    CUDA_CHECK(err, "copy to seg_d");

    show_mem_usage();

    gold_segsort(key, val, n, seg, m);

    // cout << "key:\n"; for(auto k: key) cout << k << ", "; cout << endl;
    // cout << "val:\n"; for(auto v: val) cout << v << ", "; cout << endl;

    // for(int i = 0; i < 3; i++) // test repeated execution
    bb_segsort(key_d, val_d, n, seg_d, m);

    vector<key_t> key_h(n, 0);
    vector<val_t> val_h(n, 0.0);
    err = hipMemcpy(&key_h[0], key_d, sizeof(key_t)*n, hipMemcpyDeviceToHost);
    CUDA_CHECK(err, "copy from key_d");
    err = hipMemcpy(&val_h[0], val_d, sizeof(val_t)*n, hipMemcpyDeviceToHost);
    CUDA_CHECK(err, "copy from val_d");

    // cout << "key_h:\n"; for(auto k: key_h) cout << k << ", "; cout << endl;
    // cout << "val_h:\n"; for(auto v: val_h) cout << v << ", "; cout << endl;

    index_t cnt = 0;
    for(index_t i = 0; i < n; i++)
        if(key[i] != key_h[i]) cnt++;
    if(cnt != 0) printf("[NOT PASSED] checking keys: #err = %i (%4.2f%% #nnz)\n", cnt, 100.0*(double)cnt/n);
    else printf("[PASSED] checking keys\n");
    cnt = 0;
    for(index_t i = 0; i < n; i++)
        if(val[i] != val_h[i]) cnt++;
    if(cnt != 0) printf("[NOT PASSED] checking vals: #err = %i (%4.2f%% #nnz)\n", cnt, 100.0*(double)cnt/n);
    else printf("[PASSED] checking vals\n");

    err = hipFree(key_d);
    CUDA_CHECK(err, "free key_d");
    err = hipFree(val_d);
    CUDA_CHECK(err, "free val_d");
    err = hipFree(seg_d);
    CUDA_CHECK(err, "free seg_d");
}


template<class K, class T>
void gold_segsort(vector<K> &key, vector<T> &val, index_t n, const vector<seg_t> &seg, index_t m)
{
    vector<pair<K,T>> pairs;
    for(index_t i = 0; i < n; i++)
    {
        pairs.push_back({key[i], val[i]});
    }

    for(index_t i = 0; i < m; i++)
    {
        seg_t st = seg[i];
        seg_t ed = (i<m-1)?seg[i+1]:n;
        stable_sort(pairs.begin()+st, pairs.begin()+ed, [&](pair<K,T> a, pair<K,T> b){ return a.first < b.first;});
        // sort(pairs.begin()+st, pairs.begin()+ed, [&](pair<K,T> a, pair<K,T> b){ return a.first < b.first;});
    }

    for(index_t i = 0; i < n; i++)
    {
        key[i] = pairs[i].first;
        val[i] = pairs[i].second;
    }
}

int show_mem_usage()
{
    hipError_t err;
     // show memory usage of GPU
    size_t free_byte ;
    size_t total_byte ;
    err = hipMemGetInfo(&free_byte, &total_byte);
    CUDA_CHECK(err, "check memory info.");
    size_t used_byte  = total_byte - free_byte;
    printf("GPU memory usage: used = %4.2lf MB, free = %4.2lf MB, total = %4.2lf MB\n",
        used_byte/1024.0/1024.0, free_byte/1024.0/1024.0, total_byte/1024.0/1024.0);
    return hipSuccess;
}
